#include <iostream>
#include <hip/hip_runtime.h>

#define N 2  // Very small matrix for simplicity

__global__ void matrixMul(int *A, int *B, int *C, int width) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum = 0;

    for (int i = 0; i < width; i++)
        sum += A[row * width + i] * B[i * width + col];

    C[row * width + col] = sum;
}

int main() {
    int size = N * N * sizeof(int);
    int h_A[N*N] = {1, 2, 3, 4};
    int h_B[N*N] = {5, 6, 7, 8};
    int h_C[N*N];

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); hipMalloc(&d_B, size); hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    matrixMul<<<1, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Resultant Matrix C:" << std::endl;
    for (int i = 0; i < N*N; i++) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % N == 0) std::cout << std::endl;
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}

// nvcc vector_add.cu -o vector_add
// ./vector_add