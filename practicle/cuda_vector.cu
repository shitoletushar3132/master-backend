#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

int main() {
    int n = 1 << 20; // 1 million elements
    size_t size = n * sizeof(float);

    float *h_A = new float[n];
    float *h_B = new float[n];
    float *h_C = new float[n];

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "First 5 results: ";
    for (int i = 0; i < 5; i++)
        std::cout << h_C[i] << " ";
    std::cout << std::endl;

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;

    return 0;
}


// nvcc vector_add.cu -o vector_add
// ./vector_add